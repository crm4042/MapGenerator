#include "hip/hip_runtime.h"
#include "RandomHelper.cuh"
#include <stdio.h>

__global__ void seedRandomizer(hiprandState** cs, int xDim, int yDim, int r) {
	//Gets the thread numbers
	int threadX = threadIdx.x + blockIdx.x * blockDim.x;
	int threadY = threadIdx.y + blockIdx.y * blockDim.y;

	//Gets the stride
	int strideX = gridDim.x*blockDim.x;
	int strideY = gridDim.y*blockDim.y;

	//Loops through the array seeding the randomizer
	for (int y = threadY; y < yDim; y += strideY) {
		for (int x = threadX; x < xDim; x += strideX) {
			//Seeds the randomizer
			hiprand_init(1234, r+y + x * xDim, 0, &cs[y][x]);
		}
	}
}

hiprandState** getCS(int statesX, int statesY) {
	hiprandState** cs;

	//Allocates the matrix for curandStates
	hipMallocManaged(&cs, statesY * sizeof(hiprandState*));

	for (int y = 0; y < statesY; y++) {
		hipMallocManaged(&cs[y], statesX * sizeof(hiprandState));
	}

	return cs;
}

void freeCS(hiprandState** cs, int statesX, int statesY) {
	for (int y = 0; y < statesY; y++) {
		hipFree(&cs[y]);
	}
	hipFree(&cs);
}